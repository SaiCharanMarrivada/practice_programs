
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void fill_ones(int *array, int size) {
    if (array[threadIdx.x] < size) array[threadIdx.x] = 1;
}

int main() {
    int size = 256;
    int *d_array;
    hipMalloc(&d_array, size * sizeof(int));
    fill_ones<<<1, 256>>>(d_array, size);
    int h_array[256];

    hipMemcpy(h_array, d_array, sizeof(int) * size, hipMemcpyDeviceToHost);
    for (int i = 0; i < 256; i++) {
        printf("%d ", h_array[i]);

        if (i % 16 == 15) printf("\n");
    }
}
