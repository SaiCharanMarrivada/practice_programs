
#include <hip/hip_runtime.h>
#include <cstdio>

#define THREADS_PER_BLOCK 512

__global__ void kernel_copy(int *d_output, int *d_input, size_t size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < size) d_output[id] = d_input[id];
}

#define BENCHMARK(n)                                                      \
    float *bench_copy##n(int nelements, int repetitions) {                \
        int nblocks = nelements / THREADS_PER_BLOCK;                      \
        if (nblocks % THREADS_PER_BLOCK) nblocks++;                       \
        hipEvent_t start, stop;                                          \
        hipEventCreate(&start);                                          \
        hipEventCreate(&stop);                                           \
        int *d_output, *d_input;                                          \
        hipMalloc(&d_output, nelements * sizeof(int));                   \
        hipMalloc(&d_input, nelements * sizeof(int));                    \
        float *timings = (float *)malloc(sizeof(float) * repetitions);    \
        float time_elapsed;                                               \
        for (int i = 0; i < repetitions; i++) {                           \
            hipEventRecord(start, 0);                                    \
            CODE##n;                                                      \
            hipEventRecord(stop, 0);                                     \
            hipEventSynchronize(stop);                                   \
            hipEventElapsedTime(&time_elapsed, start, stop);             \
            timings[i] = time_elapsed;                                    \
        }                                                                 \
        hipFree(d_input);                                                \
        hipFree(d_output);                                               \
        return timings;                                                   \
    }

#define CODE1 hipMemcpy(d_output, d_input, nelements * sizeof(int), \
                hipMemcpyDeviceToDevice)
#define CODE2 kernel_copy<<<nblocks, THREADS_PER_BLOCK>>>(d_output, d_input, nelements)

BENCHMARK(1);
BENCHMARK(2);

int main() {
    int nelements = (1 << 27) / sizeof(int);
    float *timings1 = bench_copy1(nelements, 10);
    float *timings2 = bench_copy2(nelements, 10);

    printf("timings1: ");
    for (int i = 0; i < 10; i++) {
        printf("%lf, ", timings1[i]);
    }
    printf("\n");

    printf("timings2: ");
    for (int i = 0; i < 10; i++) {
        printf("%lf, ", timings2[i]);
    }
    printf("\n");

    free(timings1);
    free(timings2);

}
